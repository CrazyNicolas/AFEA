#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
//Using thrust HPC libarary to High Level Programming
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>

#define THREADNUM 4
#define BLOCKNUM 2
#define LENGTH 16
__global__ void kernel(float *arr)
{
	arr[threadIdx.x] = arr[threadIdx.x] + threadIdx.x;
}
__global__ void add(int *a, int *b, int *c, int num)
{
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

__global__ void reduce(int *a, int *b)
{
	int tid = threadIdx.x;
	__shared__ int sData[16];
	sData[tid] = a[tid];
	__syncthreads();
	int i = 16;
	while(i >= 1){
		i/=2;
		if(tid < i ){
			sData[tid] = sData[tid] + sData[tid + i];
		}
		__syncthreads();
	}

	if( tid == 0){
		b[tid] = sData[tid];
	}
}

__global__ void dot_production(int *a, int *b, int *r)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int totalId = tid + bid * blockDim.x;
	int period  = BLOCKNUM * THREADNUM
	__shared__ int sData[THREADNUM];
	sData[tid] = 0;
	while(totalId < LENGTH){
		sData[tid] += a[totalId] * b[totalId];
		totalId += period;
	}
	__syncthreads();

	//reduce
	for(int i = THREADNUM/2; i>0; i/=2){
		if(tid < i) sData[tid] = sData[tid] + sData[tid+i];
		__syncthreads();
	}

	if(tid == 0){
		r[bid] = sData[0];
	}

}

void cpuSum(int *a, int *b){
	b[0] = 0;
	for(int i = 0; i < 16; i++){
		b[0] += a[i];
	}
}
int main(int argc, char **argv)
{
	////basic opration
	// hipSetDevice(0);
	// float *aGPU;
	// hipMalloc((void**)&aGPU,16*sizeof(float));
	// float a[16] = {0};
	// hipMemcpy(aGPU,a,16*sizeof(float),hipMemcpyHostToDevice);
	// kernel<<<1,16>>>(aGPU);
	// hipMemcpy(a,aGPU,16*sizeof(float),hipMemcpyDeviceToHost);
	// for(int i = 0; i <16; i++)
	// {
	// 	printf("%.2f\t",a[i]);
	// }
	// hipFree(aGPU);
	// hipDeviceReset();
	// printf("\n");
	// int gpuCount = -1;
	// hipGetDeviceCount(&gpuCount);
	// printf("This PC has %d GPUs\n",gpuCount);

	// //look up some information of GPU
	// hipDeviceProp_t prop;
	// hipGetDeviceProperties(&prop,0);
	// printf("max thread per block: %d\n",prop.maxThreadsPerBlock);
	// printf("Total memory: %zd\n",prop.totalConstMem);
	
	//// add operation
	// hipSetDevice(0);
	// const int num = 10;
	// int a[num],b[num],c[num];
	// int *a_gpu, *b_gpu, *c_gpu;
	// for(int i = 0 ; i < num; i++)
	// {
	// 	a[i] = i;
	// 	b[i] = i * i;
	// }
	// hipMalloc((void**)&a_gpu,sizeof(int) * num);
	// hipMalloc((void**)&b_gpu,sizeof(int) * num);
	// hipMalloc((void**)&c_gpu,sizeof(int) * num);

	// //copy data
	// hipMemcpy(a_gpu, a, num * sizeof(int), hipMemcpyHostToDevice);
	// hipMemcpy(b_gpu, b, num * sizeof(int), hipMemcpyHostToDevice);

	// //do
	// add<<<1,num>>>(a_gpu,b_gpu,c_gpu,num);

	// // get data
	// hipMemcpy(c, c_gpu, num * sizeof(int), hipMemcpyDeviceToHost);

	// // io
	// for(int i = 0; i < num; i++){
	// 	printf("%d\n",c[i]);
	// }


	// Reduce运算
	// const int num = 16;
	// int a[num];
	// int b[1];
	// int *aGpu;
	// int *bGpu;
	// for(int i = 0 ; i < num; i++){
	// 	a[i] = i * (i+1);
	// }
	// hipMalloc((void**)&aGpu,num*sizeof(int));
	// hipMalloc((void**)&bGpu, 1*sizeof(int));
	// hipMemcpy(aGpu, a, num*sizeof(int), hipMemcpyHostToDevice);

	// //do
	// reduce<<<1,num>>>(aGpu,bGpu);
	// hipMemcpy(b, bGpu, sizeof(int), hipMemcpyDeviceToHost);
	// printf("sum = %d\n",b[0]);

	// //comparing performance of GPU and CPU
	// // run 10000 times to calculate the total time
	// clock_t startTime,endTime;
	// startTime = clock();
	// for(int i = 0; i < 1000000; i++){
	// 	reduce<<<1,num>>>(aGpu,bGpu);
	// }
	// endTime = clock();
	// printf("1000000 times on GPU need: %.3f seconds\n",
	// 	(double)(endTime-startTime)/CLOCKS_PER_SEC);

	// // cpu case
	// startTime = clock();
	// for(int i = 0; i < 1000000; i++){
	// 	cpuSum(a,b);
	// }
	// endTime = clock();
	// printf("1000000 times on CPU need: %.3f seconds\n",
	// 	(double)(endTime-startTime)/CLOCKS_PER_SEC);
	// hipFree(aGpu);
	// hipFree(bGpu);

	//create data on host
	// thrust::host_vector<int> h_vec(20);
	// thrust::generate(h_vec.begin(),h_vec.end(),rand);
	// for(int i = 0; i < h_vec.size();i++)
	// {
	// 	printf("h[%d] = %d\t",i,h_vec[i]);
	// }
	// printf("\n");
	// //transfer host data to device
	// thrust::device_vector<int> d_vec = h_vec;

	// //sort on GPU
	// thrust::sort(d_vec.begin(),d_vec.end());

	// // transfer back to host
	// thrust::copy(d_vec.begin(),d_vec.end(),h_vec.begin());
	// for(int i = 0; i < h_vec.size();i++)
	// {
	// 	printf("h[%d] = %d\t",i,h_vec[i]);
	// }
	// printf("\n");

	// different BLOCK how to cooperate
	int a[LENGTH],b[LENGTH];
	for(int i = 0; i < LENGTH; i++){
		a[i] = i * (i+1);
		b[i] = i * (i-2);
	}
	int *aGPU, *bGpu;
	hipMalloc((void**)&aGpu, sizeof(int) * LENGTH);
	hipMemcpy(aGpu, a, sizeof(int) * LENGTH, hipMemcpyHostToDevice);
	hipMalloc((void**)&bGpu, sizeof(int) * LENGTH);
	hipMemcpy(bGpu, b, sizeof(int) * LENGTH, hipMemcpyHostToDevice);

	return 0;
}