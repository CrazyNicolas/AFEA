
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
__global__ void kernel(float *arr)
{
	arr[threadIdx.x] = arr[threadIdx.x] + threadIdx.x;
}
__global__ void add(int *a, int *b, int *c, int num)
{
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

__global__ void reduce(int *a, int *b)
{
	int tid = threadIdx.x;
	__shared__ int sData[16];
	sData[tid] = a[tid];
	__syncthreads();
	int i = 16;
	while(i >= 1){
		i/=2;
		if(tid < i ){
			sData[tid] = sData[tid] + sData[tid + i];
		}
		__syncthreads();
	}

	if( tid == 0){
		b[tid] = sData[tid];
	}
}

void cpuSum(int *a, int *b){
	b[0] = 0;
	for(int i = 0; i < 16; i++){
		b[0] += a[i];
	}
}
int main(int argc, char **argv)
{
	////basic opration
	// cudaSetDevice(0);
	// float *aGPU;
	// cudaMalloc((void**)&aGPU,16*sizeof(float));
	// float a[16] = {0};
	// cudaMemcpy(aGPU,a,16*sizeof(float),cudaMemcpyHostToDevice);
	// kernel<<<1,16>>>(aGPU);
	// cudaMemcpy(a,aGPU,16*sizeof(float),cudaMemcpyDeviceToHost);
	// for(int i = 0; i <16; i++)
	// {
	// 	printf("%.2f\t",a[i]);
	// }
	// cudaFree(aGPU);
	// cudaDeviceReset();
	// printf("\n");
	// int gpuCount = -1;
	// cudaGetDeviceCount(&gpuCount);
	// printf("This PC has %d GPUs\n",gpuCount);

	// //look up some information of GPU
	// cudaDeviceProp prop;
	// cudaGetDeviceProperties(&prop,0);
	// printf("max thread per block: %d\n",prop.maxThreadsPerBlock);
	// printf("Total memory: %zd\n",prop.totalConstMem);
	
	//// add operation
	// cudaSetDevice(0);
	// const int num = 10;
	// int a[num],b[num],c[num];
	// int *a_gpu, *b_gpu, *c_gpu;
	// for(int i = 0 ; i < num; i++)
	// {
	// 	a[i] = i;
	// 	b[i] = i * i;
	// }
	// cudaMalloc((void**)&a_gpu,sizeof(int) * num);
	// cudaMalloc((void**)&b_gpu,sizeof(int) * num);
	// cudaMalloc((void**)&c_gpu,sizeof(int) * num);

	// //copy data
	// cudaMemcpy(a_gpu, a, num * sizeof(int), cudaMemcpyHostToDevice);
	// cudaMemcpy(b_gpu, b, num * sizeof(int), cudaMemcpyHostToDevice);

	// //do
	// add<<<1,num>>>(a_gpu,b_gpu,c_gpu,num);

	// // get data
	// cudaMemcpy(c, c_gpu, num * sizeof(int), cudaMemcpyDeviceToHost);

	// // io
	// for(int i = 0; i < num; i++){
	// 	printf("%d\n",c[i]);
	// }


	// Reduce运算
	const int num = 16;
	int a[num];
	int b[1];
	int *aGpu;
	int *bGpu;
	for(int i = 0 ; i < num; i++){
		a[i] = i * (i+1);
	}
	hipMalloc((void**)&aGpu,num*sizeof(int));
	hipMalloc((void**)&bGpu, 1*sizeof(int));
	hipMemcpy(aGpu, a, num*sizeof(int), hipMemcpyHostToDevice);

	//do
	reduce<<<1,num>>>(aGpu,bGpu);
	hipMemcpy(b, bGpu, sizeof(int), hipMemcpyDeviceToHost);
	printf("sum = %d\n",b[0]);

	//comparing performance of GPU and CPU
	// run 10000 times to calculate the total time
	clock_t startTime,endTime;
	startTime = clock();
	for(int i = 0; i < 1000000; i++){
		reduce<<<1,num>>>(aGpu,bGpu);
	}
	endTime = clock();
	printf("1000000 times on GPU need: %.3f seconds\n",
		(double)(endTime-startTime)/CLOCKS_PER_SEC);

	// cpu case
	startTime = clock();
	for(int i = 0; i < 1000000; i++){
		cpuSum(a,b);
	}
	endTime = clock();
	printf("1000000 times on CPU need: %.3f seconds\n",
		(double)(endTime-startTime)/CLOCKS_PER_SEC);
	hipFree(aGpu);
	hipFree(bGpu);
	return 0;
}