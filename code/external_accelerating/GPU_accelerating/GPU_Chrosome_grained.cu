#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<thrust/device_vector.h>
// Cellular Model, so we define a grad like group
#define BLOCK_COL 10
#define BLOCK_ROW 10
#define THREADS_PER_BLOCK 52
// Some constants for Genetic operation
#define MUTATION_PROBABILITY 0.300
int Rand(int X, int Y){
	return rand()%(Y-X+1) + X;
}
void init_group(int arr[][THREADS_PER_BLOCK]){
	int r,temp;
	for(int bid = 0; bid < BLOCK_COL * BLOCK_ROW; bid++){
		for(int i = 0; i < THREADS_PER_BLOCK; i++) arr[bid][i] = i;
		for(int i = 0; i < THREADS_PER_BLOCK; i++){
			r = Rand(0,THREADS_PER_BLOCK-1);
			temp = arr[bid][i];
			arr[bid][i] = arr[bid][r];
			arr[bid][r] = temp;
		}
		// for(int i = 1; i <=THREADS_PER_BLOCK; i++){
		// 	printf("%3d ",arr[bid][i-1]);
		// 	if(i % 10 == 0) printf("\n");
		// }
		//printf("\n");
	}
}

__global__ void crossover(int *parents, int *offsprings, unsigned int *randoms, int pitch){
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	// define 2 crossover points 
	__shared__ int mother,a,b;
	if(tid == 0){
		mother = randoms[bid * 3]%4; // up down right left
		a = randoms[bid * 3 + 1]%THREADS_PER_BLOCK;
		b = randoms[bid * 3 + 2]%THREADS_PER_BLOCK;
	}
	__syncthreads();
	int *parent = (int*)((char*)parents + bid * pitch);
	int *offspring = (int*)((char*)offsprings + bid * pitch);
	offspring[tid] = parent[tid];
	__syncthreads();
	if(a > b){
		int temp;
		temp = a;
		a = b;
		b = temp;
	}

	// cellular model, so the bound need to be considered 
	if(tid >= a && tid <= b){
		if(mother == 0){// up
			int *parent_prime = (int*)((char*)parents + ((bid -1 + BLOCK_ROW)%BLOCK_ROW) * pitch);
			offspring[tid] = parent_prime[tid];
		}
		if(mother == 1){//down
			int *parent_prime = (int*)((char*)parents + ((bid +1)%BLOCK_ROW) * pitch);
			offspring[tid] = parent_prime[tid];
		}
		if(mother == 2){//left
			offspring[tid] = parent[(tid- 1+ BLOCK_COL)%BLOCK_COL];
		}
		if(mother == 3){//right
			offspring[tid] = parent[(tid +1)%BLOCK_COL];
		}
	}
	
}

// randomly choose 2 vertices and swag them
__global__ void mutation(int *group, unsigned int *randoms, float *pms,int pitch){
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	__shared__ int a,b;
	__shared__ float pm;
	a = randoms[bid * 2];
	b = randoms[bid * 2 + 1];
	pm = pms[bid];
	if(tid == a && pm <= MUTATION_PROBABILITY){
		int *individual = (int*)((char*)group + bid * pitch);
		int temp = individual[a];
		individual[a] = individual[b];
		individual[b] = temp;
	}
}

// calculate fitness, actually a reduction operation
__global__ void calculate_fitness(int *group, float *DIS, float *immediates, 
	int pitch, int pitch_dis){
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	__shared__ int *individual;
	individual=(int*)((char*)group + bid * pitch);
	__shared__ float *immediate;
	immediate = (float*)((char*)immediates + bid * pitch);
	__syncthreads();
	float *DIS_row = (float*)((char*)DIS + individual[tid] * pitch_dis);
	// boundary case
	if(tid == (THREADS_PER_BLOCK-1)){
		immediate[tid] = DIS_row[individual[0]];
	}else{
		immediate[tid] = DIS_row[individual[tid+1]];
	}
}

__global__ void reduce_fitness(float *immediates, float *fitness, int pitch){
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	__shared__ float *individual;
	individual=(float*)((char*)immediates + bid * pitch);
	__syncthreads();
	for(int i = blockDim.x/2; i > 0; i/=2){//todo:  fixup 2^n set rhem as 0
		if(tid < i){
			individual[tid] = individual[tid] + individual[tid + i];
		} 
	}
	__syncthreads();
	if(tid == 0){
		fitness[bid] = individual[0];
	}
}


// using REDUCTION to calculate fitness then Natural Selection
__global__ void select_offspring(int *parents, int *offsprings, 
	float* f_parents, float *f_offspring, int pitch){
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	__shared__ int *parent;
	__shared__ int *offspring;
	parent = (int*)((char*)parents + bid * pitch);
	offspring = (int*)((char*)offsprings + bid * pitch);
	__syncthreads();
	if(f_parents[bid] < f_offspring[bid]){
		parent[tid] = offspring[tid];
	}
}
int main(){
	srand((unsigned)time(NULL));

	//init Distance Matrix, This peocedure can also be written as a kernel!!!
	float DIS[THREADS_PER_BLOCK][THREADS_PER_BLOCK];
	for(int i = 0; i < THREADS_PER_BLOCK; i++){
		for(int j = 0; j < THREADS_PER_BLOCK; j++){
			//For convenience to TEST set 0
			DIS[i][j] = 0;
			//TODO: calculate from real data
		}
	}
	//allocate DM to GPU
	size_t pitch_dis;
	float *DIS_gpu;
	hipMallocPitch((void**)&DIS_gpu, &pitch_dis, 
		THREADS_PER_BLOCK * sizeof(float), THREADS_PER_BLOCK);
	hipMemcpy2D(DIS_gpu,pitch_dis,DIS,THREADS_PER_BLOCK*sizeof(float), 
		THREADS_PER_BLOCK*sizeof(float), THREADS_PER_BLOCK, hipMemcpyHostToDevice);

	//init a group on CPU
	int group[BLOCK_COL*BLOCK_ROW][THREADS_PER_BLOCK];
	init_group(group);
	int *group_gpu;
	size_t pitch;

	// allocate this group on GPU
	hipMallocPitch((void**)&group_gpu,&pitch,THREADS_PER_BLOCK * sizeof(int),
		BLOCK_COL*BLOCK_ROW);
	hipMemcpy2D(group_gpu, pitch, group, THREADS_PER_BLOCK * sizeof(int), 
		THREADS_PER_BLOCK*sizeof(int), BLOCK_ROW*BLOCK_COL, hipMemcpyHostToDevice);
	printf("pitch = %d\n",(int)pitch);

	// allocate offspring(same size with group) 
	int *offsprings_gpu;
	hipMallocPitch((void**)&offsprings_gpu, &pitch, THREADS_PER_BLOCK * sizeof(int), 
		BLOCK_ROW * BLOCK_COL);

	//allocate immediates matrix on GPU
	float *immediates_gpu;
	hipMallocPitch((void**)&immediates_gpu,&pitch, THREADS_PER_BLOCK * sizeof(float), 
		BLOCK_ROW * BLOCK_COL);
	hipMemset2D(immediates_gpu, pitch, 0, pitch, BLOCK_ROW * BLOCK_COL);

	// allocate fitness of individuals in group on CPU
	//float fitness_parents[BLOCK_ROW * BLOCK_COL];
	//float fitness_offsprings[BLOCK_ROW * BLOCK_COL];
	// allocate fitness of individuals in group on GPU
	float *fitness_parents_gpu;
	hipMalloc((void**)&fitness_parents_gpu, BLOCK_ROW*BLOCK_COL*sizeof(float));
	float *fitness_offsprings_gpu;
	hipMalloc((void**)&fitness_offsprings_gpu, BLOCK_ROW*BLOCK_COL*sizeof(float));

	// prepare a DEVICE random valueble generator
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_XORWOW);

	//devlaration and allcation of DEVICE rvs
	unsigned int *crossover_rv_gpu;
	unsigned int *mutation_location_gpu;
	float *mutation_rv_gpu;
	hipMalloc((void**)&crossover_rv_gpu, 
		3 * BLOCK_ROW * BLOCK_COL * sizeof(unsigned int));
	hipMalloc((void**)&mutation_location_gpu, 
		2 * BLOCK_ROW * BLOCK_COL * sizeof(unsigned int));
	hipMalloc((void**)&mutation_rv_gpu, BLOCK_ROW*BLOCK_COL*sizeof(float));


	//start iterating process
	int epoch = 500;
	time_t start = clock(); //calculating time
	while(--epoch){
		// firstly, generate all neccessary rvs for crossover
		hiprandGenerate(gen,crossover_rv_gpu,3*BLOCK_ROW*BLOCK_COL);
		// do cross over
		crossover<<<BLOCK_ROW * BLOCK_COL, THREADS_PER_BLOCK>>>(group_gpu,offsprings_gpu,
			crossover_rv_gpu, pitch);// 500 epoch only take 0.005s to execute

		// generate all necessary rvs for mutation
		hiprandGenerate(gen,mutation_location_gpu,2*BLOCK_ROW*BLOCK_COL);
		hiprandGenerateUniform(gen,mutation_rv_gpu,BLOCK_ROW*BLOCK_COL);// 0-1 uniform
		mutation<<<BLOCK_ROW * BLOCK_COL, THREADS_PER_BLOCK>>>(offsprings_gpu, 
			mutation_location_gpu, mutation_rv_gpu,pitch);// 500 epoch only take 0.001s to execute

		//calculate fitness
		calculate_fitness<<<BLOCK_ROW*BLOCK_COL,THREADS_PER_BLOCK>>>(group_gpu,DIS_gpu,immediates_gpu,pitch,pitch_dis);
		reduce_fitness<<<BLOCK_ROW*BLOCK_COL,pitch/sizeof(float)>>>(immediates_gpu,fitness_parents_gpu,pitch);
		//thrust is slow
		// for(int i= 0; i < BLOCK_ROW * BLOCK_COL; i++){
		// 	float *immediate_gpu = (float*)((char*)immediates_gpu + i * pitch);
		// 	thrust::device_vector<float> dvec(immediate_gpu,immediate_gpu + THREADS_PER_BLOCK);
			
		// 	fitness_parents[i] = thrust::reduce(dvec.begin(),dvec.end(),(float)0, thrust::plus<float>());
		// }
		calculate_fitness<<<BLOCK_ROW*BLOCK_COL,THREADS_PER_BLOCK>>>(offsprings_gpu,DIS_gpu,immediates_gpu,pitch,pitch_dis);
		reduce_fitness<<<BLOCK_ROW*BLOCK_COL,pitch/sizeof(float)>>>(immediates_gpu,fitness_offsprings_gpu,pitch);
		// for(int i= 0; i < BLOCK_ROW * BLOCK_COL; i++){
		// 	float *immediate_gpu = (float*)((char*)immediates_gpu + i * pitch);
		// 	thrust::device_vector<float> dvec(immediate_gpu,immediate_gpu+THREADS_PER_BLOCK);
			
		// 	fitness_offsprings[i] = thrust::reduce(dvec.begin(),dvec.end(),(float)0, thrust::plus<float>());
		// }
		
		select_offspring<<<BLOCK_ROW * BLOCK_COL, THREADS_PER_BLOCK>>>(group_gpu,offsprings_gpu,
			fitness_parents_gpu,fitness_offsprings_gpu,pitch);

		// data_recording<<<BLOCK_ROW * BLOCK_COL, THREADS_PER_BLOCK>>>();
	}
	time_t end = clock();
	double time_used = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Time Used: %.5f; Time Used Per Epoch: %.5f\n",time_used,time_used/500);
	hipFree(group_gpu);
	hipFree(offsprings_gpu);
	hipFree(crossover_rv_gpu);
	hipFree(mutation_rv_gpu);
	return 0;
}

