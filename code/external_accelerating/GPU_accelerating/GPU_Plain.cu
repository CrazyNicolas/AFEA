#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hiprand.h>
#include<iostream>
#include<hiprand/hiprand_kernel.h>
using namespace std;

/*in this paper each thread are responsible for one chorosome*/
#define NUM_OF_POPULATION 20
#define CITES 14
#define Pc 0.9
#define Pm 0.3
#define Ps 0.8

#define EPOCH 500
//read TSP data from *.tsp file
double** Read_TSP(char* path)
{
	unsigned int a;
	freopen(path, "r", stdin);
	double** res = new double* [NUM_OF_POPULATION];
	double* x = new double[CITES];
	double* y = new double[CITES];
	for (unsigned int i = 0; i < CITES; i++)
	{
		res[i] = new double[CITES];
		std::cin >> a;	//Some benchmark data have additional index values before coordinates.
		std::cin >> x[i] >> y[i];
	}
	fclose(stdin);
	for (unsigned int i = 0; i < CITES; i++)
		for (unsigned int j = 0; j < CITES; j++)
			res[i][j] = res[j][i] = sqrt((x[i] - x[j]) * (x[i] - x[j]) + (y[i] - y[j]) * (y[i] - y[j]));
	delete[] x, y;
	return res;
}

unsigned int Rand(unsigned int X, unsigned int Y){
	return rand()%(Y-X+1) + X;
}
void init_group(unsigned int arr[][CITES]){
	unsigned int r,temp;
	for(unsigned int bid = 0; bid < NUM_OF_POPULATION; bid++){
		for(unsigned int i = 0; i < CITES; i++) arr[bid][i] = i;
		for(unsigned int i = 0; i < CITES; i++){
			r = Rand(0,CITES-1);
			temp = arr[bid][i];
			arr[bid][i] = arr[bid][r];
			arr[bid][r] = temp;
		}
		// for(unsigned int i = 1; i <=CITES; i++){
		// 	printf("%3d ",arr[bid][i-1]);
		// 	if(i % 10 == 0) printf("\n");
		// }
		//printf("\n");
	}
}


__global__ void get_fitness(unsigned int *Group, double *dis, double* fitness, unsigned int pitch, unsigned int pitch_dis){
	unsigned int tid = threadIdx.x;
	unsigned int *individual = (unsigned int*)((char*)Group + tid*pitch);
	double tour_dis = 0.0;
	for(unsigned int i = 0; i < CITES; i++){
		//printf("%d\n",individual[i]);
		double *dis_temp = (double*)((char*)dis + individual[i] * pitch_dis);
		if(i == CITES -1){
			//printf("%.2lf\n",dis_temp[0]);
			tour_dis += dis_temp[0];
		}else{
			//printf("%.2lf\n",dis_temp[i+1]);
			tour_dis += dis_temp[individual[i+1]];
		}
	}
	fitness[tid] = tour_dis;
}

__global__ void get_offsprings(unsigned int *parents, unsigned int *offsprings, double *dis,
	unsigned int *location_crossover, float *rv_crossover,
	unsigned int *location_mutation, float *rv_mutation, 
	double *fitness_offsprings, unsigned int pitch, unsigned int pitch_dis){
	/*each thread responsible for generating one offspring*/
	unsigned int tid = threadIdx.x;
	//crossover
	unsigned int *parent = (unsigned int*)((char*)parents + tid*pitch);
	unsigned int *offspring = (unsigned int*)((char*)offsprings + tid*pitch);
	unsigned int cl = location_crossover[tid]%CITES;
	if(rv_crossover[tid] < Pc){
		for(unsigned int i = 0; i < CITES; i++){
			offspring[i] = parent[(cl+i)%CITES];
		}
	}else{
		for(unsigned int i = 0; i < CITES; i++){
			offspring[i] = parent[i];
		}
	}

	// mutation
	if(rv_mutation[tid] < Pm){
		unsigned int a = location_mutation[tid * 2]%CITES;
		unsigned int b = location_mutation[tid * 2 + 1]%CITES;
		unsigned int temp = offspring[a];
		offspring[a] = offspring[b];
		offspring[b] = temp;
	}

	// get fitness
	double tour_dis = 0.0;
	for(unsigned int i = 0; i < CITES; i++){
		//printf("%d\n",individual[i]);
		double *dis_temp = (double*)((char*)dis + offspring[i] * pitch_dis);
		if(i == CITES -1){
			//printf("%.2lf\n",dis_temp[0]);
			tour_dis += dis_temp[0];
		}else{
			//printf("%.2lf\n",dis_temp[i+1]);
			tour_dis += dis_temp[offspring[i+1]];
		}
	}
	fitness_offsprings[tid] = tour_dis;

}

__global__ void select(unsigned int *parents, unsigned int *offsprings, 
	unsigned int *candidate_select, float *rv_select, 
	double *fitness_parents, double *fitness_offsprings,unsigned int pitch){
	int tid = threadIdx.x;
	unsigned int temp[CITES];
	double fitness_temp;
	unsigned int pidx = candidate_select[tid*2]%NUM_OF_POPULATION;
	unsigned int oidx = candidate_select[tid*2+1]%NUM_OF_POPULATION;
	unsigned int *parent = (unsigned int*)((char*)parents + pidx*pitch);
	unsigned int *offspring = (unsigned int*)((char*)offsprings + oidx*pitch);
	if(fitness_offsprings[oidx] > fitness_parents[pidx]){
		for(int i = 0; i < CITES; i++){
			temp[i] = parent[i];
			fitness_temp = fitness_parents[pidx];
		}
	}else{
		for(int i = 0; i < CITES; i++){
			temp[i] = offspring[i];
			fitness_temp = fitness_offsprings[oidx];
		}
	}
	__syncthreads();
	unsigned int *parent_tid = (unsigned int*)((char*)parents + tid*pitch);
	for(int i =0; i < CITES; i++){
		parent_tid[i] = temp[i];
	}
	fitness_parents[tid] = fitness_temp;
}

__global__ void print_dis(double* dis, unsigned int pitch_dis){
	for(unsigned int i = 0; i < CITES ; i++){
		double *dis_temp = (double*)((char*)dis + i*pitch_dis);
		for(unsigned int j = 0; j < CITES; j++){
			printf("%.2lf ",dis_temp[j]);
		}
		printf("\n");
	}
}

int main(unsigned int argc, char **argv){
	srand((unsigned)time(NULL));
	//init Distance Matrix
	double DIS[CITES][CITES];
	double **dis = Read_TSP((char*)"burma14.tsp");
	//test if read correctly
	for(unsigned int i = 0; i < CITES; i++){
		for(unsigned int j = 0; j < CITES; j++){
			DIS[i][j] = dis[i][j];
			cout<<DIS[i][j]<<" ";
		}
		cout<<endl;
	}

	//allocate DM to GPU
	size_t pitch_dis;
	double *DIS_gpu;
	hipMallocPitch((void**)&DIS_gpu, &pitch_dis, 
		CITES * sizeof(double), CITES);
	hipMemcpy2D(DIS_gpu,pitch_dis,DIS,CITES*sizeof(double), 
		CITES*sizeof(double), CITES, hipMemcpyHostToDevice);

	//init a group on CPU
	unsigned int group[NUM_OF_POPULATION][CITES];
	init_group(group);
	//test init
	// for(unsigned int i = 0; i < NUM_OF_POPULATION; i++){
	// 	for(unsigned int j = 0; j < CITES; j++){
	// 		cout<<group[i][j]<<" ";
	// 	}
	// 	cout<<endl;
	// }
	unsigned int *group_gpu;
	size_t pitch;

	// allocate this group on GPU
	hipMallocPitch((void**)&group_gpu,&pitch,CITES * sizeof(unsigned int),
		NUM_OF_POPULATION);
	hipMemcpy2D(group_gpu, pitch, group, CITES * sizeof(unsigned int), 
		CITES*sizeof(unsigned int), NUM_OF_POPULATION, hipMemcpyHostToDevice);
	printf("pitch = %d, dis_pitch = %d\n",(unsigned int)pitch,(unsigned int)pitch_dis);

	// allocate offsprings(same size with group) on GPU
	unsigned int *offsprings_gpu;
	hipMallocPitch((void**)&offsprings_gpu, &pitch, CITES * sizeof(unsigned int), 
		NUM_OF_POPULATION);
	// allocate offsprings on CPU
	unsigned int offsprings[NUM_OF_POPULATION][CITES];


	// allocate fitness_parent on GPU
	double *fitness_parents_gpu;
	hipMalloc((void**)&fitness_parents_gpu, NUM_OF_POPULATION*sizeof(double));
	// allocate fitness_offspring on GPU
	double *fitness_offsprings_gpu;
	hipMalloc((void**)&fitness_offsprings_gpu, NUM_OF_POPULATION*sizeof(double));

	// prepare a DEVICE random valuable generator
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_XORWOW);

	// allocate rvs shown in this paper
	unsigned int *location_crossover;
	float *rv_crossover;
	unsigned int *location_mutation;
	float *rv_mutation;
	unsigned int *candidate_select;
	float *rv_select;
	hipMalloc((void**)&location_crossover, sizeof(unsigned int) * NUM_OF_POPULATION);
	hipMalloc((void**)&rv_crossover, sizeof(float) * NUM_OF_POPULATION);
	hipMalloc((void**)&location_mutation, sizeof(unsigned int) * NUM_OF_POPULATION * 2);
	hipMalloc((void**)&rv_mutation, sizeof(float) * NUM_OF_POPULATION);
	hipMalloc((void**)&candidate_select, sizeof(unsigned int) * NUM_OF_POPULATION * 2);
	hipMalloc((void**)&rv_select, sizeof(float) * NUM_OF_POPULATION);

	//main iteration
	//kernel 1
	get_fitness<<<1,NUM_OF_POPULATION>>>(group_gpu,DIS_gpu,
		fitness_parents_gpu,pitch,pitch_dis);
	int epoch = EPOCH;
	while(epoch--){
		//kernel 2
		hiprandGenerate(gen,location_crossover,NUM_OF_POPULATION);
		hiprandGenerateUniform(gen,rv_crossover,NUM_OF_POPULATION);
		hiprandGenerate(gen,location_mutation,2*NUM_OF_POPULATION);
		hiprandGenerateUniform(gen,rv_mutation,NUM_OF_POPULATION);
		get_offsprings<<<1,NUM_OF_POPULATION>>>(group_gpu,offsprings_gpu,DIS_gpu,
			location_crossover,rv_crossover,location_mutation,rv_mutation,
			fitness_offsprings_gpu,pitch,pitch_dis);
		hiprandGenerate(gen,candidate_select,2*NUM_OF_POPULATION);
		hiprandGenerateUniform(gen,rv_select,NUM_OF_POPULATION);
		select<<<1,NUM_OF_POPULATION>>>(group_gpu,offsprings_gpu,candidate_select,
			rv_mutation,fitness_parents_gpu,fitness_offsprings_gpu,pitch);

	}
	//DEBUG-START
	/* test get_fitness()*/
	// get_fitness<<<1,NUM_OF_POPULATION>>>(group_gpu,DIS_gpu,
	// 	fitness_parents_gpu,pitch,pitch_dis);
	// double fitness[NUM_OF_POPULATION];
	// hipMemcpy(fitness, fitness_parents_gpu, NUM_OF_POPULATION*sizeof(double), 
	// 	hipMemcpyDeviceToHost);
	// for(unsigned int i = 0; i < NUM_OF_POPULATION; i++){
	// 	cout<<fitness[i]<<" ";
	// }
	/* test dis successfully tranfered to GPU*/
	//print_dis<<<1,1>>>(DIS_gpu,pitch_dis);

	/*test get_offsprings()*/
	// hiprandGenerate(gen,location_crossover,NUM_OF_POPULATION);
	// hiprandGenerateUniform(gen,rv_crossover,NUM_OF_POPULATION);
	// hiprandGenerate(gen,location_mutation,2*NUM_OF_POPULATION);
	// hiprandGenerateUniform(gen,rv_mutation,NUM_OF_POPULATION);
	// get_offsprings<<<1,NUM_OF_POPULATION>>>(group_gpu,offsprings_gpu,DIS_gpu,
	// 	location_crossover,rv_crossover,location_mutation,rv_mutation,
	// 	fitness_offsprings_gpu,pitch,pitch_dis);

	// hipMemcpy2D(offsprings, CITES*sizeof(unsigned int), offsprings_gpu, pitch, 
	// 	CITES*sizeof(unsigned int), NUM_OF_POPULATION, hipMemcpyDeviceToHost);
	// for(int i =0 ; i < NUM_OF_POPULATION; i++){
	// 	for(int j = 0 ; j < CITES; j++){
	// 		cout<<offsprings[i][j]<<" ";
	// 	}
	// 	cout<<endl;
	// }
	// double fitness[NUM_OF_POPULATION];
	// hipMemcpy(fitness, fitness_offsprings_gpu, NUM_OF_POPULATION*sizeof(double), 
	// 	hipMemcpyDeviceToHost);
	// for(unsigned int i = 0; i < NUM_OF_POPULATION; i++){
	// 	cout<<fitness[i]<<" ";
	// }
	/*test select*/
	// hiprandGenerate(gen,candidate_select,2*NUM_OF_POPULATION);
	// hiprandGenerateUniform(gen,rv_select,NUM_OF_POPULATION);
	// select<<<1,NUM_OF_POPULATION>>>(group_gpu,offsprings_gpu,candidate_select,
	// 	rv_mutation,fitness_parents_gpu,fitness_offsprings_gpu,pitch);
	hipMemcpy2D(offsprings, CITES*sizeof(unsigned int),group_gpu, pitch, 
		CITES*sizeof(unsigned int), NUM_OF_POPULATION, hipMemcpyDeviceToHost);
	for(int i =0 ; i < NUM_OF_POPULATION; i++){
		for(int j = 0 ; j < CITES; j++){
			cout<<offsprings[i][j]<<" ";
		}
		cout<<endl;
	}
	double fitness[NUM_OF_POPULATION];
	hipMemcpy(fitness, fitness_parents_gpu, NUM_OF_POPULATION*sizeof(double), 
		hipMemcpyDeviceToHost);
	for(unsigned int i = 0; i < NUM_OF_POPULATION; i++){
		cout<<fitness[i]<<" ";
	}

	//DEBUG-END

	return 0;
}