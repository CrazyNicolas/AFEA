
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hiprand/hiprand.h>
#include<iostream>
#include<hiprand/hiprand_kernel.h>
#include<algorithm>
using namespace std;
// Cellular Model, so we define a grid like group
#define BLOCK_COL 10
#define BLOCK_ROW 10
#define THREADS_PER_BLOCK 52
// Some constants for Genetic operation
#define MUTATION_PROBABILITY 0.300

//SBX const
#define ETA 2

//define a struct for sorting with key
typedef struct{
	float value;
	int key;
}SortObj;
bool compare(SortObj a, SortObj b){
	return a.value < b.value;
}

//read TSP data from *.tsp file
double** Read_TSP(char* path)
{
	unsigned int a;
	freopen(path, "r", stdin);
	double** res = new double* [BLOCK_ROW*BLOCK_COL];
	double* x = new double[THREADS_PER_BLOCK];
	double* y = new double[THREADS_PER_BLOCK];
	for (unsigned int i = 0; i < THREADS_PER_BLOCK; i++)
	{
		res[i] = new double[THREADS_PER_BLOCK];
		std::cin >> a;	//Some benchmark data have additional index values before coordinates.
		std::cin >> x[i] >> y[i];
	}
	fclose(stdin);
	for (unsigned int i = 0; i < THREADS_PER_BLOCK; i++)
		for (unsigned int j = 0; j < THREADS_PER_BLOCK; j++)
			res[i][j] = res[j][i] = sqrt((x[i] - x[j]) * (x[i] - x[j]) + (y[i] - y[j]) * (y[i] - y[j]));
	delete[] x, y;
	return res;
}


unsigned int Rand(unsigned int X, unsigned int Y){
	return rand()%(Y-X+1) + X;
}
void init_group(unsigned int arr[][THREADS_PER_BLOCK]){
	unsigned int r,temp;
	for(unsigned int bid = 0; bid < BLOCK_COL * BLOCK_ROW; bid++){
		for(unsigned int i = 0; i < THREADS_PER_BLOCK; i++) arr[bid][i] = i;
		for(unsigned int i = 0; i < THREADS_PER_BLOCK; i++){
			r = Rand(0,THREADS_PER_BLOCK-1);
			temp = arr[bid][i];
			arr[bid][i] = arr[bid][r];
			arr[bid][r] = temp;
		}
		// for(unsigned int i = 1; i <=THREADS_PER_BLOCK; i++){
		// 	printf("%3d ",arr[bid][i-1]);
		// 	if(i % 10 == 0) printf("\n");
		// }
		//printf("\n");
	}
}

// this crossover are not suitable for TSP instead refer to SBX_crossover below
__global__ void crossover(unsigned int *parents, unsigned int *offsprings, unsigned int *randoms, unsigned int pitch){
	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	// define 2 crossover points 
	__shared__ unsigned int mother,a,b;
	if(tid == 0){
		mother = randoms[bid * 3]%4; // up down right left
		a = randoms[bid * 3 + 1]%THREADS_PER_BLOCK;
		b = randoms[bid * 3 + 2]%THREADS_PER_BLOCK;
	}
	__syncthreads();
	unsigned int *parent = (unsigned int*)((char*)parents + bid * pitch);
	unsigned int *offspring = (unsigned int*)((char*)offsprings + bid * pitch);
	offspring[tid] = parent[tid];
	__syncthreads();
	if(a > b){
		unsigned int temp;
		temp = a;
		a = b;
		b = temp;
	}

	// cellular model, so the bound need to be considered 
	if(tid >= a && tid <= b){
		if(mother == 0){// up
			unsigned int *parent_prime = (unsigned int*)((char*)parents + ((bid -1 + BLOCK_ROW)%BLOCK_ROW) * pitch);
			offspring[tid] = parent_prime[tid];
		}
		if(mother == 1){//down
			unsigned int *parent_prime = (unsigned int*)((char*)parents + ((bid +1)%BLOCK_ROW) * pitch);
			offspring[tid] = parent_prime[tid];
		}
		if(mother == 2){//left
			offspring[tid] = parent[(tid- 1+ BLOCK_COL)%BLOCK_COL];
		}
		if(mother == 3){//right
			offspring[tid] = parent[(tid +1)%BLOCK_COL];
		}
	}
	
}

//SBX
__global__ void SBX_crossover(unsigned int *parents, float *pre_offsprings, 
	unsigned int *randoms, float* random_SBX, unsigned int pitch){
	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	__shared__ unsigned int mother;
	__shared__ float beta;
	if(tid == 0){
		mother = randoms[bid]%4; // up down right left
		if(random_SBX[bid] > 0.5){
			beta = pow( (double)2 * random_SBX[bid] , (double)1 / (1 + ETA));
		}else{
			beta = pow((double)1/(2 - 2*random_SBX[bid]), (double)1/(1+ETA));
		}
	}
	__syncthreads();
	unsigned int *parent = (unsigned int*)((char*)parents + bid * pitch);
	unsigned int *pre_offspring = (unsigned int*)((char*)pre_offsprings + bid * pitch);
	if(mother == 0){// up
		unsigned int *parent_prime = (unsigned int*)((char*)parents + (((bid-BLOCK_ROW + BLOCK_ROW*BLOCK_COL)/BLOCK_ROW)%BLOCK_ROW*BLOCK_ROW + bid%BLOCK_ROW) * pitch);
		pre_offspring[tid] = 0.5 * ((1+beta) * parent_prime[tid] + (1-beta) * parent[tid]);
	}
	if(mother == 1){//down
		unsigned int *parent_prime = (unsigned int*)((char*)parents + ((((bid+BLOCK_ROW)/BLOCK_ROW)%BLOCK_ROW)*BLOCK_ROW + bid%BLOCK_ROW) * pitch);
		pre_offspring[tid] = 0.5 * ((1+beta) * parent_prime[tid] + (1-beta) * parent[tid]);
	}
	if(mother == 2){//left
		unsigned int *parent_prime = (unsigned int*)((char*)parents + (bid/BLOCK_ROW*BLOCK_ROW + (bid -1)%BLOCK_ROW) * pitch);
		pre_offspring[tid] = 0.5 * ((1+beta) * parent_prime[tid] + (1-beta) * parent[tid]);
	}
	if(mother == 3){//right
		unsigned int *parent_prime = (unsigned int*)((char*)parents + (bid/BLOCK_ROW*BLOCK_ROW + (bid +1)%BLOCK_ROW) * pitch);
		pre_offspring[tid] = 0.5 * ((1+beta) * parent_prime[tid] + (1-beta) * parent[tid]);
	}
	

}

// randomly choose 2 vertices and swag them
__global__ void mutation(unsigned int *group, unsigned int *randoms, float *pms,unsigned int pitch){
	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	__shared__ unsigned int a,b;
	__shared__ float pm;
	a = randoms[bid * 2] % THREADS_PER_BLOCK;
	b = randoms[bid * 2 + 1] %THREADS_PER_BLOCK;
	pm = pms[bid];
	if(tid == a && pm <= MUTATION_PROBABILITY){
		unsigned int *individual = (unsigned int*)((char*)group + bid * pitch);
		unsigned int temp = individual[a];
		individual[a] = individual[b];
		individual[b] = temp;
	}
}

// calculate fitness, actually a reduction operation
__global__ void calculate_fitness(unsigned int *group, double *DIS, double *immediates, 
	unsigned int pitch, unsigned int pitch_dis){
	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	__shared__ unsigned int *individual;
	individual=(unsigned int*)((char*)group + bid * pitch);
	__shared__ double *immediate;
	immediate = (double*)((char*)immediates + bid * pitch);
	__syncthreads();
	double *DIS_row = (double*)((char*)DIS + individual[tid] * pitch_dis);
	// boundary case
	if(tid == (THREADS_PER_BLOCK-1)){
		immediate[tid] = DIS_row[individual[0]];
	}else{
		//printf("%.2lf\n",DIS_row[individual[tid+1]]);
		immediate[tid] = DIS_row[individual[tid+1]];
	}
}

__global__ void reduce_fitness(double *immediates, double *fitness, unsigned int pitch){
	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	__shared__ double *individual;
	individual=(double*)((char*)immediates + bid * pitch);
	__syncthreads();
	for(unsigned int i = blockDim.x/2; i > 0; i/=2){
		if(tid < i){
			individual[tid] = individual[tid] + individual[tid + i];
		} 
	}
	__syncthreads();
	if(tid == 0){
		fitness[bid] = individual[0];
		//printf("%.2lf\n",individual[0]);
	}
}


// using REDUCTION to calculate fitness then Natural Selection
__global__ void select_offspring(unsigned int *parents, unsigned int *offsprings, 
	double* f_parents, double *f_offspring, unsigned int pitch){
	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	__shared__ unsigned int *parent;
	__shared__ unsigned int *offspring;
	parent = (unsigned int*)((char*)parents + bid * pitch);
	offspring = (unsigned int*)((char*)offsprings + bid * pitch);
	__syncthreads();
	if(f_parents[bid] > f_offspring[bid]){
		parent[tid] = offspring[tid];
	}
	__syncthreads();
	if(f_parents[bid] > f_offspring[bid]){
		if(tid == 0) f_parents[bid] = f_offspring[bid];
	}
}
int main(){
	srand((unsigned)time(NULL));
	//init Distance Matrix, This peocedure can also be written as a kernel!!!
	double DIS[THREADS_PER_BLOCK][THREADS_PER_BLOCK];
	double **dis = Read_TSP((char*)"berlin52.tsp");
	//test if read correctly
	for(unsigned int i = 0; i < THREADS_PER_BLOCK; i++){
		for(unsigned int j = 0; j < THREADS_PER_BLOCK; j++){
			DIS[i][j] = dis[i][j];
			cout<<DIS[i][j]<<" ";
		}
		cout<<endl;
	}

	//allocate DM to GPU
	size_t pitch_dis;
	double *DIS_gpu;
	hipMallocPitch((void**)&DIS_gpu, &pitch_dis, 
		THREADS_PER_BLOCK * sizeof(double), THREADS_PER_BLOCK);
	hipMemcpy2D(DIS_gpu,pitch_dis,DIS,THREADS_PER_BLOCK*sizeof(double), 
		THREADS_PER_BLOCK*sizeof(double), THREADS_PER_BLOCK, hipMemcpyHostToDevice);

	//init a group on CPU
	unsigned int group[BLOCK_COL*BLOCK_ROW][THREADS_PER_BLOCK];
	init_group(group);
	//test init
	// for(unsigned int i = 0; i < BLOCK_ROW * BLOCK_COL; i++){
	// 	for(unsigned int j = 0; j < THREADS_PER_BLOCK; j++){
	// 		cout<<group[i][j]<<" ";
	// 	}
	// 	cout<<endl;
	// }
	unsigned int *group_gpu;
	size_t pitch;

	// allocate this group on GPU
	hipMallocPitch((void**)&group_gpu,&pitch,THREADS_PER_BLOCK * sizeof(unsigned int),
		BLOCK_COL*BLOCK_ROW);
	hipMemcpy2D(group_gpu, pitch, group, THREADS_PER_BLOCK * sizeof(unsigned int), 
		THREADS_PER_BLOCK*sizeof(unsigned int), BLOCK_ROW*BLOCK_COL, hipMemcpyHostToDevice);
	printf("pitch = %d, dis_pitch = %d\n",(unsigned int)pitch,(unsigned int)pitch_dis);

	// allocate pre_offsprings in CPU
	float pre_offsprings[BLOCK_ROW*BLOCK_COL][THREADS_PER_BLOCK];
	// allocate pre_offspring in GPU
	float *pre_offsprings_gpu;
	hipMallocPitch((void**)&pre_offsprings_gpu, &pitch, THREADS_PER_BLOCK * sizeof(unsigned int), 
		BLOCK_ROW * BLOCK_COL);
	// allocate offsprings(same size with group) on GPU
	unsigned int *offsprings_gpu;
	hipMallocPitch((void**)&offsprings_gpu, &pitch, THREADS_PER_BLOCK * sizeof(unsigned int), 
		BLOCK_ROW * BLOCK_COL);
	// allocate offsprings on CPU
	unsigned int offsprings[BLOCK_ROW*BLOCK_COL][THREADS_PER_BLOCK];

	//allocate immediates matrix on GPU
	double *immediates_gpu;
	hipMallocPitch((void**)&immediates_gpu,&pitch, THREADS_PER_BLOCK * sizeof(double), 
		BLOCK_ROW * BLOCK_COL);
	hipMemset2D(immediates_gpu, pitch, 0, pitch, BLOCK_ROW * BLOCK_COL);

	// allocate fitness of individuals in group on CPU
	//double fitness_parents[BLOCK_ROW * BLOCK_COL];
	//double fitness_offsprings[BLOCK_ROW * BLOCK_COL];
	// allocate fitness of individuals in group on GPU
	double *fitness_parents_gpu;
	hipMalloc((void**)&fitness_parents_gpu, BLOCK_ROW*BLOCK_COL*sizeof(double));
	double *fitness_offsprings_gpu;
	hipMalloc((void**)&fitness_offsprings_gpu, BLOCK_ROW*BLOCK_COL*sizeof(double));

	// prepare a DEVICE random valuable generator
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_XORWOW);

	//devlaration and allocation of DEVICE rvs
	unsigned int *crossover_rv_gpu;
	float *crossover_rv_SBX_gpu;
	unsigned int *mutation_location_gpu;
	float *mutation_rv_gpu;
	hipMalloc((void**)&crossover_rv_gpu, 
		BLOCK_ROW * BLOCK_COL * sizeof(unsigned int));
	hipMalloc((void**)&crossover_rv_SBX_gpu, BLOCK_ROW*BLOCK_COL*sizeof(float));
	hipMalloc((void**)&mutation_location_gpu, 
		2 * BLOCK_ROW * BLOCK_COL * sizeof(unsigned int));
	hipMalloc((void**)&mutation_rv_gpu, BLOCK_ROW*BLOCK_COL*sizeof(float));

	//define struct 2D array for sorting pre_offsprings in crossover_SBX
	SortObj **sobjs = new SortObj*[BLOCK_ROW*BLOCK_COL];
	for(int i = 0; i < BLOCK_ROW * BLOCK_COL; i++){
		sobjs[i] = new SortObj[THREADS_PER_BLOCK];
	}

	//start iterating process
	unsigned int epoch = 500;
	time_t start = clock(); //calculating time
	while(--epoch){
		// firstly, generate all neccessary rvs for crossover
		hiprandGenerate(gen,crossover_rv_gpu,BLOCK_ROW*BLOCK_COL);
		hiprandGenerateUniform(gen,crossover_rv_SBX_gpu,BLOCK_ROW*BLOCK_COL);// 0-1 uniform
		// do cross over
		SBX_crossover<<<BLOCK_ROW * BLOCK_COL, THREADS_PER_BLOCK>>>(group_gpu,pre_offsprings_gpu,
			crossover_rv_gpu, crossover_rv_SBX_gpu, pitch);// 500 epoch only take 0.005s to execute
		//now we got SBX result: pre_offsprings sort on cpu and copy to offsprings_gpu
		//1. copy out pre_offsprings 
		hipMemcpy2D(pre_offsprings, THREADS_PER_BLOCK*sizeof(float), 
			pre_offsprings_gpu, pitch, THREADS_PER_BLOCK*sizeof(float), BLOCK_ROW*BLOCK_COL, hipMemcpyDeviceToHost);
		//2. construct sorting obj
		for(int i = 0; i < BLOCK_ROW * BLOCK_COL; i++){
			for(int j = 0; j < THREADS_PER_BLOCK; j++){
				sobjs[i][j].value = pre_offsprings[i][j];
				sobjs[i][j].key = j;
			}
		}
		//3. sorting
		for(int i=0; i<BLOCK_ROW*BLOCK_COL; i++){
			sort(sobjs[i],sobjs[i]+THREADS_PER_BLOCK,compare);
			for(int j =0; j <THREADS_PER_BLOCK; j++){//set offsprings
				offsprings[i][sobjs[i][j].key] = j;
			}
		}

		//4. copy offsprings to offsprings_gpu
		hipMemcpy2D(offsprings_gpu, pitch, offsprings, THREADS_PER_BLOCK*sizeof(float)
			, THREADS_PER_BLOCK*sizeof(float), BLOCK_ROW*BLOCK_COL, hipMemcpyHostToDevice);
		// generate all necessary rvs for mutation
		hiprandGenerate(gen,mutation_location_gpu,2*BLOCK_ROW*BLOCK_COL);
		hiprandGenerateUniform(gen,mutation_rv_gpu,BLOCK_ROW*BLOCK_COL);// 0-1 uniform
		mutation<<<BLOCK_ROW * BLOCK_COL, THREADS_PER_BLOCK>>>(offsprings_gpu, 
			mutation_location_gpu, mutation_rv_gpu,pitch);// 500 epoch only take 0.001s to execute

		//calculate fitness
		calculate_fitness<<<BLOCK_ROW*BLOCK_COL,THREADS_PER_BLOCK>>>(group_gpu,DIS_gpu,immediates_gpu,pitch,pitch_dis);
		reduce_fitness<<<BLOCK_ROW*BLOCK_COL,pitch/sizeof(double)>>>(immediates_gpu,fitness_parents_gpu,pitch);
		//thrust is slow
		// for(unsigned int i= 0; i < BLOCK_ROW * BLOCK_COL; i++){
		// 	double *immediate_gpu = (double*)((char*)immediates_gpu + i * pitch);
		// 	thrust::device_vector<double> dvec(immediate_gpu,immediate_gpu + THREADS_PER_BLOCK);
			
		// 	fitness_parents[i] = thrust::reduce(dvec.begin(),dvec.end(),(double)0, thrust::plus<double>());
		// }
		calculate_fitness<<<BLOCK_ROW*BLOCK_COL,THREADS_PER_BLOCK>>>(offsprings_gpu,DIS_gpu,immediates_gpu,pitch,pitch_dis);
		reduce_fitness<<<BLOCK_ROW*BLOCK_COL,pitch/sizeof(double)>>>(immediates_gpu,fitness_offsprings_gpu,pitch);
		// for(unsigned int i= 0; i < BLOCK_ROW * BLOCK_COL; i++){
		// 	double *immediate_gpu = (double*)((char*)immediates_gpu + i * pitch);
		// 	thrust::device_vector<double> dvec(immediate_gpu,immediate_gpu+THREADS_PER_BLOCK);
			
		// 	fitness_offsprings[i] = thrust::reduce(dvec.begin(),dvec.end(),(double)0, thrust::plus<double>());
		// }
		
		select_offspring<<<BLOCK_ROW * BLOCK_COL, THREADS_PER_BLOCK>>>(group_gpu,offsprings_gpu,
			fitness_parents_gpu,fitness_offsprings_gpu,pitch);

		

		
	}
	time_t end = clock();
	double time_used = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Time Used: %.5f; Time Used Per Epoch: %.5f\n",time_used,time_used/500);

	//in the end, find the best individual and his fitness
	double result[BLOCK_ROW * BLOCK_COL];
	hipMemcpy(result, fitness_parents_gpu, 
		BLOCK_COL*BLOCK_ROW*sizeof(double), hipMemcpyDeviceToHost);
	for(unsigned int i =0; i < BLOCK_ROW; i++){
		for(unsigned int j = 0; j < BLOCK_COL; j++){
			printf("%.2lf\t",result[i * BLOCK_ROW +j]);
		}
		printf("\n");
	}
	printf("\n");
	hipMemcpy2D(group, THREADS_PER_BLOCK * sizeof(unsigned int), group_gpu, pitch, 
		THREADS_PER_BLOCK*sizeof(unsigned int), BLOCK_ROW*BLOCK_COL, hipMemcpyDeviceToHost);

	for(unsigned int i =0; i <BLOCK_ROW * BLOCK_COL; i++){
		for(unsigned int j = 0; j < THREADS_PER_BLOCK; j++){
			printf("%d ",group[i][j]);
		}

		printf("\n\n");
	}

	//DEBUG-START
	// calculate_fitness<<<BLOCK_ROW*BLOCK_COL,THREADS_PER_BLOCK>>>(group_gpu,
	// 	DIS_gpu,immediates_gpu,pitch,pitch_dis);
	// reduce_fitness<<<BLOCK_ROW*BLOCK_COL,pitch/sizeof(double)>>>(immediates_gpu,
	// 		fitness_parents_gpu,pitch);
	// double res[BLOCK_ROW * BLOCK_COL];
	// cudaMemcpy(res, fitness_parents_gpu,
	//  BLOCK_ROW*BLOCK_COL*sizeof(double), cudaMemcpyDeviceToHost);
	// for(unsigned int i =0; i < BLOCK_ROW; i++){
	// 	for(unsigned int j = 0; j < BLOCK_COL; j++){
	// 		printf("%.2lf\t",res[i * BLOCK_ROW +j]);
	// 	}
	// 	printf("\n");
	// }

	//DEBUG-END
	hipFree(group_gpu);
	hipFree(offsprings_gpu);
	hipFree(crossover_rv_gpu);
	hipFree(mutation_rv_gpu);
	return 0;
}

